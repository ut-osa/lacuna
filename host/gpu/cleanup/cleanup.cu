#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
	
	void** device_mem[2000];
	int alloced[2000];
	void* device=NULL;
	long toAlloc;
	int i;
	long totalAlloc=0;
	for (i=0;i<1600;i++){
		float alloc=1.5;
		bool success=false;
		while(alloc>0){
			toAlloc=alloc*1024*1024;
			if (hipMalloc((void**)&device_mem[i],toAlloc)!=hipSuccess){
			alloc-=0.1;
			fprintf(stderr,"%ld %s\n",toAlloc,hipGetErrorString(hipPeekAtLastError()));
			}else{
				success=true;
				totalAlloc+=toAlloc;
				alloced[i]=toAlloc;
				break;
				
			}
		}
		if(success==false) break;
	}
	
	fprintf(stderr,"Allocated %ld GB of memory\n",totalAlloc );
	fprintf(stderr,"Cleaning\n");
	for(int z=0;z<=i;z++){
		fprintf(stderr,"%d %p\n",z,device_mem[z]);
		if (hipMemset(device_mem[z],0,alloced[z])!=hipSuccess){
			printf("Memory cannot be accessed -> %z??\n");
		}
	}
//	cudaFree(device);
	return 0;
}

