#include "hip/hip_runtime.h"

#include <stdarg.h>
#include <cutil.h>
#include <stdio.h>
#include <stdlib.h>
extern "C"{
#include "aes_gpu.h"
#include "timer.h"
}
#include <stdlib.h>

#define _THREADS_PER_BLOCK 256
const int MaxThreadsPerBlock = _THREADS_PER_BLOCK;




/* Global array : step for optimization*/
uchar* d_expkey  = NULL; 
uchar* d_sbox  = NULL; 
uchar* d_invsbox  = NULL; 
uchar* d_xtimee  = NULL; 
uchar* d_xtimeb  = NULL; 
uchar* d_xtimed  = NULL; 
uchar* d_xtime9  = NULL; 
uchar* d_in_enc  = NULL; 
uchar* d_out_enc = NULL; 
uchar* d_in_dec  = NULL; 
uchar* d_out_dec = NULL; 

uchar* h_in_enc, *h_out_enc;
uchar* h_in_dec, *h_out_dec;



int gpu_fd=-1;
texture<unsigned char, 1, hipReadModeElementType> tex_sbox;
texture<unsigned char, 1, hipReadModeElementType> tex_invsbox;
#define SBOX(i)         ((unsigned char)tex1Dfetch(tex_sbox,(i)))
#define INVSBOX(i)         ((unsigned char)tex1Dfetch(tex_invsbox,(i)))

texture<unsigned char, 1, hipReadModeElementType> tex_xtimee;
texture<unsigned char, 1, hipReadModeElementType> tex_xtimeb;
texture<unsigned char, 1, hipReadModeElementType> tex_xtimed;
texture<unsigned char, 1, hipReadModeElementType> tex_xtime9;
#define XTIMEE(i)         ((unsigned char)tex1Dfetch(tex_xtimee,(i)))
#define XTIMEB(i)         ((unsigned char)tex1Dfetch(tex_xtimeb,(i)))
#define XTIMED(i)         ((unsigned char)tex1Dfetch(tex_xtimed,(i)))
#define XTIME9(i)         ((unsigned char)tex1Dfetch(tex_xtime9,(i)))

/* The global data */
__device__ __constant__ uchar _Sbox[256];
__device__ __constant__ uchar _InvSbox[256];
__device__ __constant__ uchar _Xtime2Sbox[256];
__device__ __constant__ uchar _Xtime3Sbox[256];
__device__ __constant__ uchar _Xtime2[256];
__device__ __constant__ uchar _Xtime9[256];
__device__ __constant__ uchar _XtimeB[256];
__device__ __constant__ uchar _XtimeD[256];
__device__ __constant__ uchar _XtimeE[256];

/* GPU Functions */
void _ShiftRows (uchar *state);
void _InvShiftRows (uchar *state);
void _MixSubColumns (uchar *state);
void _InvMixSubColumns (uchar *state);
void _AddRoundKey (unsigned *state, unsigned *key);
void _Encrypt (uchar *in, uchar *expkey, uchar *out);
void _Decrypt (uchar *in, uchar *expkey, uchar *out);
	



//prefetch 
union Int4Char{
	uint i32;
	uchar i8[4];
};
//#define SBOX(x) _Sbox[(x)];
#define FETCH_COL(out,input,col) (out)=((uint*)input)[(col)];
#define SAVE_COL(out,input,col) ((uint*)out)[(col)]=(input);

__device__ struct bigint gpu_bigint_plus(const struct bigint& x, uint y){
        char i=0;                                                              
        char carry=0;                                                          
	struct bigint res;
        res.d[0]=x.d[0];
        res.d[1]=x.d[1];                                                       
        res.d[2]=x.d[2];
        res.d[3]=x.d[3];
	uchar* v=(uchar*)res.d;                                 
	for(int j=y;j>0;j-=MAXVAL){                      
		uchar tmp_y=j>MAXVAL?MAXVAL:j;
	        for(i=15;i>=0;i--){
        	       if (MAXVAL != v[i] && MAXVAL- carry - v[i] >= tmp_y) {         
	                        v[i]=v[i]+tmp_y+carry;
	                        carry=0;                                               
	                        break;
        	        }                                                              
	                v[i]=tmp_y-(MAXVAL+1-v[i]-carry);
	                tmp_y=0;
	                carry=1;                                                       
        	}
	}
	return res;

}

__device__ void _ShiftRows_opt (uchar* input){
	Int4Char col0,col1,col2,col3;
	Int4Char tmp;
	FETCH_COL(col0.i32, input,0);
	FETCH_COL(col1.i32, input,1);
	FETCH_COL(col2.i32, input,2);
	FETCH_COL(col3.i32, input,3);
	
// r0	
	col0.i8[0]=SBOX(col0.i8[0]);
	col1.i8[0]=SBOX(col1.i8[0]);
	col2.i8[0]=SBOX(col2.i8[0]);
	col3.i8[0]=SBOX(col3.i8[0]);
	
// r1	

	tmp.i8[1]=SBOX(col0.i8[1]);
	col0.i8[1]=SBOX(col1.i8[1]);
	col1.i8[1]=SBOX(col2.i8[1]);
	col2.i8[1]=SBOX(col3.i8[1]);
	col3.i8[1]=tmp.i8[1];

// r2	
	tmp.i8[2]=SBOX(col0.i8[2]);
	tmp.i8[3]=SBOX(col1.i8[2]);
	col0.i8[2]=SBOX(col2.i8[2]);
	col1.i8[2]=SBOX(col3.i8[2]);
	col2.i8[2]=tmp.i8[2];
	col3.i8[2]=tmp.i8[3];
// r3
	tmp.i8[3]=SBOX(col3.i8[3]);
	col3.i8[3]=SBOX(col2.i8[3]);
	col2.i8[3]=SBOX(col1.i8[3]);
	col1.i8[3]=SBOX(col0.i8[3]);
	col0.i8[3]=tmp.i8[3];

	SAVE_COL(input,col0.i32,0);
	SAVE_COL(input,col1.i32,1);
	SAVE_COL(input,col2.i32,2);
	SAVE_COL(input,col3.i32,3);
}

/*
// exchanges columns in each of 4 rows
// row0 - unchanged, row1- shifted left 1, 
// row2 - shifted left 2 and row3 - shifted left 3
__device__ void _ShiftRows (uchar *state)
{
	//printf("  - ShiftRows\n");
	uchar tmp;
	

	// just substitute row 0
	state[0] = _Sbox[state[0]], state[4]  = _Sbox[state[4]];
	state[8] = _Sbox[state[8]], state[12] = _Sbox[state[12]];

	// rotate row 1
	tmp = _Sbox[state[1]], state[1] = _Sbox[state[5]];
	state[5] = _Sbox[state[9]], state[9] = _Sbox[state[13]], state[13] = tmp;

	// rotate row 2
	tmp = _Sbox[state[2]], state[2] = _Sbox[state[10]], state[10] = tmp;
	tmp = _Sbox[state[6]], state[6] = _Sbox[state[14]], state[14] = tmp;

	// rotate row 3
	tmp = _Sbox[state[15]], state[15] = _Sbox[state[11]];
	state[11] = _Sbox[state[7]], state[7] = _Sbox[state[3]], state[3] = tmp;
}

*/

__device__ void _InvShiftRows_opt (uchar* input){
	Int4Char col0,col1,col2,col3;
	Int4Char tmp;
	FETCH_COL(col0.i32, input,0);
	FETCH_COL(col1.i32, input,1);
	FETCH_COL(col2.i32, input,2);
	FETCH_COL(col3.i32, input,3);
	
// r0	
	col0.i8[0]=INVSBOX(col0.i8[0]);
	col1.i8[0]=INVSBOX(col1.i8[0]);
	col2.i8[0]=INVSBOX(col2.i8[0]);
	col3.i8[0]=INVSBOX(col3.i8[0]);
	
// r1	

	tmp.i8[1]=INVSBOX(col3.i8[1]);
	col3.i8[1]=INVSBOX(col2.i8[1]);
	col2.i8[1]=INVSBOX(col1.i8[1]);
	col1.i8[1]=INVSBOX(col0.i8[1]);
	col3.i8[1]=tmp.i8[1];

// r2	
	tmp.i8[2]=INVSBOX(col0.i8[2]);
	tmp.i8[3]=INVSBOX(col1.i8[2]);
	col0.i8[2]=INVSBOX(col2.i8[2]);
	col1.i8[2]=INVSBOX(col3.i8[2]);
	col2.i8[2]=tmp.i8[2];
	col3.i8[2]=tmp.i8[3];
// r3
	tmp.i8[3]=INVSBOX(col0.i8[3]);
	col0.i8[3]=INVSBOX(col1.i8[3]);
	col1.i8[3]=INVSBOX(col2.i8[3]);
	col2.i8[3]=INVSBOX(col0.i8[3]);
	col3.i8[3]=tmp.i8[3];

	SAVE_COL(input,col0.i32,0);
	SAVE_COL(input,col1.i32,1);
	SAVE_COL(input,col2.i32,2);
	SAVE_COL(input,col3.i32,3);
}
// restores columns in each of 4 rows
// row0 - unchanged, row1- shifted right 1, 
// row2 - shifted right 2 and row3 - shifted right 3
__device__ void _InvShiftRows (uchar *state)
{
	uchar tmp,tmp2;

	// restore row 0
	state[0] = _InvSbox[state[0]], state[4] = _InvSbox[state[4]];
	state[8] = _InvSbox[state[8]], state[12] = _InvSbox[state[12]];

	// restore row 1
	tmp = _InvSbox[state[13]], state[13] = _InvSbox[state[9]];
	state[9] = _InvSbox[state[5]], state[5] = _InvSbox[state[1]], state[1] = tmp;

	// restore row 2
	tmp = _InvSbox[state[2]]; 
	tmp2 = _InvSbox[state[6]]; 
	
	state[2] = _InvSbox[state[10]];
	state[6] = _InvSbox[state[14]]; 
       	state[10] = tmp;
	state[14] = tmp2;

	// restore row 3
	tmp = _InvSbox[state[3]], state[3] = _InvSbox[state[7]];
	state[7] = _InvSbox[state[11]], state[11] = _InvSbox[state[15]], state[15] = tmp;
}


#define xtime32(x) ((((x) >> 7) & 0x01010101) * 0x1b) ^ (((x) << 1) & 0xfefefefe)

__device__ void _InvMixSubColumns_opt(uchar* state) {
	Int4Char col0,col1,col2,col3;
	Int4Char tmp;
	FETCH_COL(col0.i32, state,0);
	FETCH_COL(col1.i32, state,1);
	FETCH_COL(col2.i32, state,2);
	FETCH_COL(col3.i32, state,3);
      
	tmp.i8[0]= INVSBOX(XTIMEE(col0.i8[0])^XTIMEB(col0.i8[1])^XTIMED(col0.i8[2])^XTIME9(col0.i8[3]));
	tmp.i8[1]= INVSBOX(XTIME9(col3.i8[0])^XTIMEE(col3.i8[1])^XTIMEB(col3.i8[2])^XTIMED(col3.i8[3]));
	tmp.i8[2]= INVSBOX(XTIMED(col2.i8[0])^XTIME9(col2.i8[1])^XTIMEE(col2.i8[2])^XTIMEB(col2.i8[3]));
	tmp.i8[3]= INVSBOX(XTIMEB(col1.i8[0])^XTIMED(col1.i8[1])^XTIME9(col1.i8[2])^XTIMEE(col1.i8[3]));
	SAVE_COL(state,tmp.i32,0);
	

	tmp.i8[0]= INVSBOX(XTIMEE(col1.i8[0])^XTIMEB(col1.i8[1])^XTIMED(col1.i8[2])^XTIME9(col1.i8[3]));
	tmp.i8[1]= INVSBOX(XTIME9(col0.i8[0])^XTIMEE(col0.i8[1])^XTIMEB(col0.i8[2])^XTIMED(col0.i8[3]));
	tmp.i8[2]= INVSBOX(XTIMED(col3.i8[0])^XTIME9(col3.i8[1])^XTIMEE(col3.i8[2])^XTIMEB(col3.i8[3]));
	tmp.i8[3]= INVSBOX(XTIMEB(col2.i8[0])^XTIMED(col2.i8[1])^XTIME9(col2.i8[2])^XTIMEE(col2.i8[3]));
	SAVE_COL(state,tmp.i32,1);

	tmp.i8[0]= INVSBOX(XTIMEE(col2.i8[0])^XTIMEB(col2.i8[1])^XTIMED(col2.i8[2])^XTIME9(col2.i8[3]));
	tmp.i8[1]= INVSBOX(XTIME9(col1.i8[0])^XTIMEE(col1.i8[1])^XTIMEB(col1.i8[2])^XTIMED(col1.i8[3]));
	tmp.i8[2]= INVSBOX(XTIMED(col0.i8[0])^XTIME9(col0.i8[1])^XTIMEE(col0.i8[2])^XTIMEB(col0.i8[3]));
	tmp.i8[3]= INVSBOX(XTIMEB(col3.i8[0])^XTIMED(col3.i8[1])^XTIME9(col3.i8[2])^XTIMEE(col3.i8[3]));
	SAVE_COL(state,tmp.i32,2);

	tmp.i8[0]= INVSBOX(XTIMEE(col3.i8[0])^XTIMEB(col3.i8[1])^XTIMED(col3.i8[2])^XTIME9(col3.i8[3]));
	tmp.i8[1]= INVSBOX(XTIME9(col2.i8[0])^XTIMEE(col2.i8[1])^XTIMEB(col2.i8[2])^XTIMED(col2.i8[3]));
	tmp.i8[2]= INVSBOX(XTIMED(col1.i8[0])^XTIME9(col1.i8[1])^XTIMEE(col1.i8[2])^XTIMEB(col1.i8[3]));
	tmp.i8[3]= INVSBOX(XTIMEB(col0.i8[0])^XTIMED(col0.i8[1])^XTIME9(col0.i8[2])^XTIMEE(col0.i8[3]));
	SAVE_COL(state,tmp.i32,3);
}

__device__ void MixColumn(Int4Char& ib)
{
        Int4Char ibTmp;
    	ibTmp.i32 = (ib.i32 >> 8) | (ib.i32 << 24);
        ibTmp.i32 = ibTmp.i32 ^ ib.i32;
        ib.i32 = ib.i32 ^ xtime32(ibTmp.i32);
        ibTmp.i32 = ibTmp.i32 ^ ((ibTmp.i32 << 16) | (ibTmp.i32 >> 16));
        ib.i32 = ib.i32 ^ ibTmp.i32;
}


__device__ void _MixColumns_opt(uchar* state){
	Int4Char col0,col1,col2,col3;

	FETCH_COL(col0.i32,state,0);
	FETCH_COL(col1.i32,state,1);
	FETCH_COL(col2.i32,state,2);
	FETCH_COL(col3.i32,state,3);

	MixColumn(col0);
	MixColumn(col1);
	MixColumn(col2);
	MixColumn(col3);
	
	SAVE_COL(state,col0.i32,0);
	SAVE_COL(state,col1.i32,1);
	SAVE_COL(state,col2.i32,2);
	SAVE_COL(state,col3.i32,3);
}


__device__ void _InvMixSubColumns (uchar *state)
{
	uchar tmp[4 * Nb];
	int i;

	// restore column 0
	tmp[0] = _XtimeE[state[0]] ^ _XtimeB[state[1]] ^ _XtimeD[state[2]] ^ _Xtime9[state[3]];
	tmp[5] = _Xtime9[state[0]] ^ _XtimeE[state[1]] ^ _XtimeB[state[2]] ^ _XtimeD[state[3]];
	tmp[10] = _XtimeD[state[0]] ^ _Xtime9[state[1]] ^ _XtimeE[state[2]] ^ _XtimeB[state[3]];
	tmp[15] = _XtimeB[state[0]] ^ _XtimeD[state[1]] ^ _Xtime9[state[2]] ^ _XtimeE[state[3]];

	// restore column 1
	tmp[4] = _XtimeE[state[4]] ^ _XtimeB[state[5]] ^ _XtimeD[state[6]] ^ _Xtime9[state[7]];
	tmp[9] = _Xtime9[state[4]] ^ _XtimeE[state[5]] ^ _XtimeB[state[6]] ^ _XtimeD[state[7]];
	tmp[14] = _XtimeD[state[4]] ^ _Xtime9[state[5]] ^ _XtimeE[state[6]] ^ _XtimeB[state[7]];
	tmp[3] = _XtimeB[state[4]] ^ _XtimeD[state[5]] ^ _Xtime9[state[6]] ^ _XtimeE[state[7]];

	// restore column 2
	tmp[8] = _XtimeE[state[8]] ^ _XtimeB[state[9]] ^ _XtimeD[state[10]] ^ _Xtime9[state[11]];
	tmp[13] = _Xtime9[state[8]] ^ _XtimeE[state[9]] ^ _XtimeB[state[10]] ^ _XtimeD[state[11]];
	tmp[2]  = _XtimeD[state[8]] ^ _Xtime9[state[9]] ^ _XtimeE[state[10]] ^ _XtimeB[state[11]];
	tmp[7]  = _XtimeB[state[8]] ^ _XtimeD[state[9]] ^ _Xtime9[state[10]] ^ _XtimeE[state[11]];

	// restore column 3
	tmp[12] = _XtimeE[state[12]] ^ _XtimeB[state[13]] ^ _XtimeD[state[14]] ^ _Xtime9[state[15]];
	tmp[1] = _Xtime9[state[12]] ^ _XtimeE[state[13]] ^ _XtimeB[state[14]] ^ _XtimeD[state[15]];
	tmp[6] = _XtimeD[state[12]] ^ _Xtime9[state[13]] ^ _XtimeE[state[14]] ^ _XtimeB[state[15]];
	tmp[11] = _XtimeB[state[12]] ^ _XtimeD[state[13]] ^ _Xtime9[state[14]] ^ _XtimeE[state[15]];
	for( i=0; i <  4*Nb; i++){
		state[i] = _InvSbox[ tmp[i] ];
	}
}

// encrypt/decrypt columns of the key
// n.b. you can replace this with
//      byte-wise xor if you wish.
__device__ void _AddRoundKey (unsigned *state, unsigned *key)
{
	for ( int i = 0; i < 4; ++i )
		state[i] ^= key[i];
}

__device__ void _DecryptOneAESBlock(uchar *in, uchar* expkey){

	_AddRoundKey( (unsigned*)(in), (unsigned*)expkey + Nr * Nb );
	_InvShiftRows( in );
	int round=Nr;
	for ( round = Nr-1; round>0; round-- )
	{
		_AddRoundKey( (unsigned*)(in), (unsigned*)expkey + round * Nb );
		_InvMixSubColumns_opt( in );
	}
	_AddRoundKey( (unsigned*)(in), (unsigned*)expkey + round * Nb );

}

__device__ void _EncryptOneAESBlock(uchar *in, uchar* expkey){
	_AddRoundKey( (unsigned*)(in), (unsigned*)expkey );
	
	int round=1;

	for( round = 1; round < Nr; round++)
	{
		_ShiftRows_opt( in );
		_MixColumns_opt(in);
		_AddRoundKey( (unsigned*)(in), (unsigned*)expkey + round*Nb );
	}

	_ShiftRows_opt( in );
	_AddRoundKey( (unsigned*)(in), (unsigned*)expkey + round*Nb );
	
}


__device__ void _ParallelXor_opt(struct bigint& buf, const struct bigint* iv){
	buf.d[0]^=iv->d[0];
	buf.d[1]^=iv->d[1];
	buf.d[2]^=iv->d[2];
	buf.d[3]^=iv->d[3];
}



 __global__ void encrypt_decrypt_counter(uchar *in, uchar* expkey, uchar *out, struct bigint iv ) 
{ 

	__shared__ uchar bufkey[176];

	uint block_start=blockIdx.x*blockDim.x;
	uint myId=threadIdx.x+block_start;
	struct bigint myiv=gpu_bigint_plus(iv,myId);

	if (threadIdx.x<44){
		((int*)bufkey)[threadIdx.x]=((int*)expkey)[threadIdx.x];
	}
	__syncthreads();
	_EncryptOneAESBlock((uchar*)myiv.d,(uchar*)bufkey);
	_ParallelXor_opt(myiv,((struct bigint*)in+myId));
	((struct bigint*)out)[myId]=myiv;
}



#define MAX_BLOCK_SIZE (1024*1024*16)


 __host__ int gpu_process_string(uchar* in, int length,  uchar** d_out, struct bigint iv, bool keep_result_in_gpu=0)
{

	if (length > MAX_BLOCK_SIZE) {
		fprintf(stderr,"Data size exceeds %d\n", MAX_BLOCK_SIZE);
		return -1;
	}

	if (length%(_THREADS_PER_BLOCK*16)) {
		fprintf(stderr,"GPU can be used only to encrypt multiple of  %d bytes\n", _THREADS_PER_BLOCK*16);
		return -1;
	}


	memcpy(h_in_enc,in, length);


	uchar* h_data=h_in_enc;
	uchar* d_data=d_in_enc;
	uchar* d_out_data=d_out_enc;

	CUDA_SAFE_CALL( hipMemcpy( (void*)d_data, (void*)h_data, length, hipMemcpyHostToDevice ) );

	uint units = length / 16;
	uint threads = (units >= MaxThreadsPerBlock) ? MaxThreadsPerBlock : (units % MaxThreadsPerBlock);
	uint blocks = units / threads;

	if(blocks == 0)	blocks = 1;
	char* no_encrypt=getenv(NO_ENCRYPT_ENV);
	if (!no_encrypt)
	{
		encrypt_decrypt_counter<<<blocks,threads,0>>>(d_data, d_expkey, d_out_data,  iv);
		hipDeviceSynchronize();
	}
	if (!keep_result_in_gpu){
		CUDA_SAFE_CALL( hipMemcpy( (void*)h_data, (void*)d_out_data, length, hipMemcpyDeviceToHost) );
		hipDeviceSynchronize();
		memcpy( *d_out,  h_data, length);
	}else{
		*d_out=d_out_data;
	}
	CUDA_SAFE_CALL(hipPeekAtLastError());

	return 0;
}

 int gpu_encrypt_string(uchar* in, int length,  uchar** d_out, struct bigint iv, bool keep_in_gpu_mem=0){
	return	gpu_process_string(in,length,d_out,iv,keep_in_gpu_mem);
}


 __host__ int gpu_decrypt_string(uchar* in, int length,  uchar** d_out, struct bigint iv, bool keep_in_gpu_mem=0){
	return	gpu_process_string(in,length,d_out,iv,keep_in_gpu_mem);
}

 __host__ void gpu_init(int argv, char** argc)
{
	
	//CUDA_SAFE_CALL( hipSetDeviceFlags 	( hipDeviceScheduleSpin	 ) );
	//CUT_DEVICE_INIT(argc, argv);

	CUDA_SAFE_CALL( hipMalloc( (void**) &d_sbox, 256));
	CUDA_SAFE_CALL( hipMemcpy( d_sbox, Sbox, 256, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL(hipBindTexture(0, tex_sbox, d_sbox ));

	CUDA_SAFE_CALL( hipMalloc( (void**) &d_invsbox, 256));
	CUDA_SAFE_CALL( hipMemcpy( d_invsbox, InvSbox, 256, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL(hipBindTexture(0, tex_invsbox, d_invsbox ));

	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xtimee, 256));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xtimeb, 256));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xtimed, 256));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xtime9, 256));
	
	CUDA_SAFE_CALL( hipMemcpy(d_xtimee , XtimeE, 256, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_xtimeb , XtimeB, 256, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_xtimed , XtimeD, 256, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_xtime9 , Xtime9, 256, hipMemcpyHostToDevice) );
	
	CUDA_SAFE_CALL(hipBindTexture(0, tex_xtimee,d_xtimee  ));
	CUDA_SAFE_CALL(hipBindTexture(0, tex_xtimeb,d_xtimeb  ));
	CUDA_SAFE_CALL(hipBindTexture(0, tex_xtimed,d_xtimed  ));
	CUDA_SAFE_CALL(hipBindTexture(0, tex_xtime9,d_xtime9  ));

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_Sbox"),       Sbox,       256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_InvSbox"),    InvSbox,    256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_Xtime2Sbox"), Xtime2Sbox, 256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_Xtime3Sbox"), Xtime3Sbox, 256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_Xtime2"),     Xtime2,     256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_Xtime9"),     Xtime9,     256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_XtimeB"),     XtimeB,     256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_XtimeD"),     XtimeD,     256 ) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "_XtimeE"),     XtimeE,     256 ) );


	  /* pre-allocating device memory */
  	CUDA_SAFE_CALL( hipMalloc((void **) &d_in_enc,  MAX_BLOCK_SIZE  ) );
	CUDA_SAFE_CALL( hipMalloc((void **) &d_out_enc, MAX_BLOCK_SIZE  ) );
   
	CUDA_SAFE_CALL( hipMalloc( (void **) &d_in_dec,  MAX_BLOCK_SIZE  ) );
	CUDA_SAFE_CALL( hipMalloc( (void **) &d_out_dec, MAX_BLOCK_SIZE ) );



	CUDA_SAFE_CALL( hipMalloc( (void **) &d_expkey, 4 * Nb * (Nr + 1) ) );

	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_in_enc,MAX_BLOCK_SIZE) );
	
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_out_enc,MAX_BLOCK_SIZE) );
	
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_in_dec,MAX_BLOCK_SIZE) );
	
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_out_dec,MAX_BLOCK_SIZE) );
}

 __host__ int gpu_device_count()
{
	int deviceCount = 0;

	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		return 0;
	}

   	
	return deviceCount;

	/*check devideQuesry api of sdk for more support*/

}

 __host__ int  gpu_getMaxThreadCount()
{
     hipDeviceProp_t deviceProp;
     hipGetDeviceProperties(&deviceProp, 0);

	  return deviceProp.maxThreadsPerBlock;
}

 __host__ void gpu_exit()
{
	if(d_in_enc)
		CUDA_SAFE_CALL( hipFree(d_in_enc) );
	if(d_out_enc)
		CUDA_SAFE_CALL( hipFree(d_out_enc) );

	if(d_in_dec)
		CUDA_SAFE_CALL( hipFree(d_in_dec) );
	if(d_out_dec)
		CUDA_SAFE_CALL( hipFree(d_out_dec) );
	if(d_expkey)
		CUDA_SAFE_CALL( hipFree(d_expkey) );
	if(d_sbox)
		CUDA_SAFE_CALL( hipFree(d_sbox) );
	if(d_invsbox)
		CUDA_SAFE_CALL( hipFree(d_invsbox) );

	if(d_xtimee)
		CUDA_SAFE_CALL( hipFree(d_xtimee) );
	if(d_xtimeb)
		CUDA_SAFE_CALL( hipFree(d_xtimeb) );
	if(d_xtimed)
		CUDA_SAFE_CALL( hipFree(d_xtimed) );
	if(d_xtime9)
		CUDA_SAFE_CALL( hipFree(d_xtime9) );
	hipHostFree(h_in_enc);
	hipHostFree(h_out_enc);
	hipHostFree(h_in_dec);
	hipHostFree(h_out_dec);
}

 __host__ int gpu_set_key(uchar* key )
{

	uchar expkey[4 * Nb * (Nr + 1)];
	ExpandKey( (uchar*)key, (uchar*)expkey );
        CUDA_SAFE_CALL( hipMemcpy( (void*)d_expkey, (void*)expkey, 4 * Nb * (Nr + 1), hipMemcpyHostToDevice ) );

	return 0;
}
