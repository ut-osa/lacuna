#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
extern "C"{
#include "encrypted-display-backend.h"
}
////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
//Texture reference and channel descriptor for image texture
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

//CUDA array descriptor
hipArray *a_Src;


int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ float lerpf(float a, float b, float c){
    return a + (b - a) * c;
}

__device__ float vecLen(float4 a, float4 b){
    return (
        (b.x - a.x) * (b.x - a.x) +
        (b.y - a.y) * (b.y - a.y) +
        (b.z - a.z) * (b.z - a.z)
    );
}

__device__ TColor make_color(float r, float g, float b, float a){
    return
        ((int)(a * 255.0f) << 24) |
        ((int)(b * 255.0f) << 16) |
        ((int)(g * 255.0f) <<  8) |
        ((int)(r * 255.0f) <<  0);
}



__global__ void Copy(
    TColor *dst,
    int imageW,
    int imageH
){
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;

    if(ix < imageW && iy < imageH){
        float4 fresult = tex2D(texImage, x, y);
        dst[imageW*(imageH-1)-imageW * iy + ix] = make_color(fresult.z, fresult.y, fresult.x,0);
    }
}

extern "C" void
cuda_Copy(TColor *d_dst, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    Copy<<<grid, threads>>>(d_dst, imageW, imageH);
}
////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////




extern "C"
hipError_t CUDA_Memcpy2TextureArray(uchar4* img,int imageW,int imageH)
{
    return hipMemcpyToArray(a_Src,0,0,img,imageW*imageH*sizeof(uchar4),hipMemcpyDeviceToDevice);
}

extern "C"
hipError_t CUDA_Bind2TextureArray()
{
    return hipBindTextureToArray(texImage, a_Src);
}

extern "C"
hipError_t CUDA_UnbindTexture()
{
    return hipUnbindTexture(texImage);
}

extern "C" 
hipError_t CUDA_MallocArray(uchar4 **h_Src, int imageW, int imageH)
{
    hipError_t error;

    error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
    error = hipMemcpyToArray(a_Src, 0, 0,
                              *h_Src, imageW * imageH * sizeof(uchar4),
                              hipMemcpyHostToDevice
                              );

    return error;
}


extern "C"
hipError_t CUDA_FreeArray()
{
    return hipFreeArray(a_Src);    
}

